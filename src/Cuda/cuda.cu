#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "improc.h"
#include "macher.h"
#include <stdio.h>
#include <chrono>
#include <hip/device_functions.h>


#include <iostream>

#define PI 3.14159265
#define NUM_STREAMS 2
#define WIDTH	640
#define HEIGHT	480


using namespace std;

void cv::gpu::mj::blur(const int rows,const int cols, const int k, unsigned char *src, unsigned char* dst){
	
	auto start1 = std::chrono::system_clock::now();
	int N = WIDTH;
	int M = HEIGHT;
	//hipSetDeviceFlags(hipDeviceMapHost);
	
	unsigned char* gpudataSrc;
	unsigned char* gpudataOut;
	unsigned char* cpudataOut;
	
	int size = sizeof(unsigned char)*rows*cols;
	
	hipHostMalloc 	((void **)&cpudataOut,size);	
	
	auto stop1 = std::chrono::system_clock::now();
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	//hipHostGetDevicePointer((void **)&gpudataOut,  (void *) dst , 0);
	hipMemcpy(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	auto stop2 = std::chrono::system_clock::now();
	dim3 threadsPerBlock(16,16);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	//auto start1 = std::chrono::system_clock::now();
	//hipDeviceSynchronize();
	auto stop3 = std::chrono::system_clock::now();
	//blur_noShare_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut );
	//hipDeviceSynchronize();
	auto stop4 = std::chrono::system_clock::now();
	//hipDeviceSynchronize();
	cout<<"1: Done!!!"<<endl;
	//blur_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut);
	cout<<"3: Done!!!"<<endl;
	//hipDeviceSynchronize();
	auto stop5 = std::chrono::system_clock::now();
	hipMemcpy(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
	auto duration1 = (std::chrono::duration_cast<std::chrono::microseconds>(stop1 - start1).count());
	auto duration2 = (std::chrono::duration_cast<std::chrono::microseconds>(stop2 - stop1).count());
	auto duration3 = (std::chrono::duration_cast<std::chrono::microseconds>(stop3 - stop2).count());
	auto duration4 = (std::chrono::duration_cast<std::chrono::microseconds>(stop4 - stop3).count());
	auto duration5 = (std::chrono::duration_cast<std::chrono::microseconds>(stop5 - stop4).count());
	auto duration =(std::chrono::duration_cast<std::chrono::microseconds>(stop5 - start1).count());
	
    //cout<<(int)duration1<<" "<<(int)duration2<<" " <<(int)duration3<<" " <<(int)duration4<<" " <<(int)duration5<<(int)duration6<< endl;
}
void cv::gpu::mj::sobel(const int rows,const int cols, unsigned char *src, unsigned char* dst, int mode){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* gpudataSrc;
	
	unsigned char* gpudataOut;
	
	const int size = sizeof(unsigned char)*rows*cols;
		
	
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	
	hipMemcpyAsync(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(32,32);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	

	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataSrc, gpudataOut, mode);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
}

unsigned char** cv::gpu::mj::initDisp(const int size){
	
	unsigned char** temps = (unsigned char**)malloc(8*sizeof(unsigned char*));
	
	int i = 0;
	
	for(i = 0; i < 8; i++){
		hipMalloc((void **)&(temps[i]), size);
	} 
	return temps;
}
/*
void cv::gpu::mj::disp(const int rows,const int cols, unsigned char *g_srcL, unsigned char *g_srcR, unsigned char* g_disp, unsigned char** temps){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* g_L_low_ext = temps[0];
	unsigned char* g_R_low_ext = temps[1];
	unsigned char* g_L_high_out = temps[2];
	unsigned char* g_R_high_out = temps[3];
	unsigned char* g_L_lowEdge = temps[4];
	unsigned char* g_R_lowEdge = temps[5];
	unsigned char* g_L_highEdge = temps[6];
	unsigned char* g_R_highEdge = temps[7];
	
	const int size = sizeof(unsigned char)*rows*cols;
	
	hipMemset(g_disp, 0, size);
	
	dim3 threadsPerBlock(16, 16);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	
	edgeDetect2x_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcL, g_L_high_out, g_L_low_ext, 30, 3);
	edgeDetect2x_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcR, g_R_high_out, g_R_low_ext, 30, 3);
	
	hipDeviceSynchronize();
	//hipMemset(g_L_lowEdge, 0, size);
	//hipMemset(g_L_low_ext, 0, size);
	//edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_low_ext, g_L_lowEdge);
	edgeTypeDetectCleanup<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_high_out, g_L_lowEdge);
	edgeTypeDetectCleanup<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_R_high_out, g_R_lowEdge);
	hipDeviceSynchronize();
	hipMemset(g_L_high_out, 0, size);
	hipMemset(g_R_high_out, 0, size);
	hipDeviceSynchronize();
	edgeTypeDetectCleanup<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_lowEdge, g_L_high_out);
	edgeTypeDetectCleanup<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_R_lowEdge, g_R_high_out);
	//hipMemset(g_L_highEdge, 0, size);
	//hipMemset(g_R_highEdge, 0, size);
	hipDeviceSynchronize();
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_high_out, g_L_highEdge);
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_R_high_out, g_R_highEdge);
	
	hipDeviceSynchronize();
	
	hipMemset(g_R_lowEdge, 0, size);
	hipMemset(g_R_low_ext, 0, size);
	dim3 threadsPerBlockDisp(48, 16);
	dim3 numbBlocksDisp(N/16, M/16); 
	
	edgeMacher<<<numbBlocksDisp, threadsPerBlockDisp>>>( rows, cols, g_L_highEdge, g_R_highEdge, g_srcL, g_srcR, g_disp);
	
	hipDeviceSynchronize();
	dim3 threadsPerBlock2(24, 24);
	dim3 numbBlocks2(N/ threadsPerBlock2.x,M/ threadsPerBlock2.y); 

	
	
	dim3 threadsPerBlockHori(WIDTH, 1);
	dim3 numbBlocksHori(N/WIDTH, M/1); 
	
	//filler<<<numbBlocksHori, threadsPerBlockHori>>>( rows, cols, g_R_low_ext, g_disp);
	//hipDeviceSynchronize();
	//blend_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_lowEdge, g_R_lowEdge, g_disp, 0.5, 1);
	
	hipMemset(g_L_low_ext, 0, size);
	hipMemset(g_R_low_ext, 0, size);
	hipMemset(g_L_high_out, 0, size);
	hipMemset(g_R_high_out, 0, size);
	hipMemset(g_L_lowEdge, 0, size);
	hipMemset(g_R_lowEdge, 0, size);
	hipMemset(g_L_highEdge, 0, size);
	hipMemset(g_R_highEdge, 0, size);
	
}

unsigned char** cv::gpu::mj::initDisp2C(const int size){
	
	unsigned char** temps = (unsigned char**)malloc(3*sizeof(unsigned char*));

	for(int i = 0; i < 3; i++){
		hipMalloc((void **)&(temps[i]), size);
	} 
	return temps;
}
__global__ void fillInitParams(unsigned int * t){
	if( threadIdx.x == 0 &&  threadIdx.y == 0){ 
		t[0 ] = 32; 
		t[1 ] = 32; 
		t[2 ] = 32; 
		t[3 ] = 31;
		t[4 ] = 30; 
		t[5 ] = 29; 
		t[6 ] = 28; 
		t[7 ] = 27;
		t[8 ] = 26; 
		t[9 ] = 25; 
		t[10] = 24; 
		t[11] = 23;
		t[12] = 22; 
		t[13] = 21; 
		t[14] = 20; 
		t[15] = 19;
		t[16] = 18; 
		t[17] = 17; 
		t[18] = 16; 
		t[19] = 17;
		t[20] = 16; 
		t[21] = 15; 
		t[22] = 16; 
		t[23] = 13; 
		t[24] = 12; 
	}

}
unsigned int** cv::gpu::mj::initDisp2I(const int size){
	
	unsigned int** temps = (unsigned int**)malloc(21*sizeof(unsigned int*));
	
	hipMalloc((void **)&(temps[0]), size/(2*2)*64);
	hipMalloc((void **)&(temps[1]), size/(4*4)*64);
	hipMalloc((void **)&(temps[2]), size/(8*8)*64);
	hipMalloc((void **)&(temps[3]), size/(16*16)*64);
	hipMalloc((void **)&(temps[4]), size/(32*32)*64);
	hipMalloc((void **)&(temps[5]), size/(2*2)*64);
	hipMalloc((void **)&(temps[6]), size/(4*4)*64);
	hipMalloc((void **)&(temps[7]), size/(8*8)*64);
	hipMalloc((void **)&(temps[8]), size/(16*16)*64);
	hipMalloc((void **)&(temps[9]), size/(32*32)*64);
	hipMalloc((void **)&(temps[10]), size/(2*2)*64);
	hipMalloc((void **)&(temps[11]), size/(4*4)*64);
	hipMalloc((void **)&(temps[12]), size/(8*8)*64);
	hipMalloc((void **)&(temps[13]), size/(16*16)*64);
	hipMalloc((void **)&(temps[14]), size/(32*32)*64);
	hipMalloc((void **)&(temps[15]), size/(2*2)*64);
	hipMalloc((void **)&(temps[16]), size/(4*4)*64);
	hipMalloc((void **)&(temps[17]), size/(8*8)*64);
	hipMalloc((void **)&(temps[18]), size/(16*16)*64);
	hipMalloc((void **)&(temps[19]), size/(32*32)*64);
	hipMalloc((void **)&(temps[20]), 32*sizeof(unsigned int));
	hipDeviceSynchronize();
	unsigned int * t = temps[20];
	
	fillInitParams<<<1, 1>>>(t);
	
	
	return temps;
}


void cv::gpu::mj::disp2(const int rows,const int cols, unsigned char *g_srcL, unsigned char *g_srcR, unsigned char* g_disp, unsigned char** tempsC, unsigned int** tempsI ){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* g_L_edge = tempsC[0];
	unsigned char* g_R_edge = tempsC[1];
	unsigned char* g_edgeMached = tempsC[2];
	
	unsigned int* g_match_2 = tempsI[0];
	unsigned int* g_match_4 = tempsI[1];
	unsigned int* g_match_8 = tempsI[2];
	unsigned int* g_match_16 = tempsI[3];
	unsigned int* g_match_32 = tempsI[4];
	unsigned int* g_match_2x = tempsI[5];
	unsigned int* g_match_4x = tempsI[6];
	unsigned int* g_match_8x = tempsI[7];
	unsigned int* g_match_16x = tempsI[8];
	unsigned int* g_match_32x = tempsI[9];
	unsigned int* g_match_2y = tempsI[10];
	unsigned int* g_match_4y = tempsI[11];
	unsigned int* g_match_8y = tempsI[12];
	unsigned int* g_match_16y = tempsI[13];
	unsigned int* g_match_32y = tempsI[14];
	unsigned int* g_match_2xy = tempsI[15];
	unsigned int* g_match_4xy = tempsI[16];
	unsigned int* g_match_8xy = tempsI[17];
	unsigned int* g_match_16xy = tempsI[18];
	unsigned int* g_match_32xy = tempsI[19];
	unsigned int* g_params = tempsI[20];
	
	const int size = sizeof(unsigned char)*rows*cols;
	
	hipMemset(g_disp, 0, size);
	hipMemset(g_L_edge, 0, size);
	hipMemset(g_R_edge, 0, size);

	hipDeviceSynchronize();
	dim3 threadsPerBlock(16, 16);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcL, g_L_edge, 50);
	//edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcR, g_R_edge, 40);
	
	hipDeviceSynchronize();
	dim3 threadsPerBlock24(24, 24);
	dim3 numbBlocks24(N/ threadsPerBlock24.x,M/ threadsPerBlock24.y); 
	//findDistance<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_L_edge, g_R_edge);
	findDistanceFast<<<numbBlocks24, threadsPerBlock24,64*64*sizeof(unsigned char) >>>(rows, cols, g_L_edge, g_R_edge);
	hipDeviceSynchronize();
	dim3 threadsPerBlockDisp2(64, 2);
	dim3 numbBlocksDisp2(N/2, M/2); 
	dim3 threadsPerBlockDisp4(64, 4);
	dim3 numbBlocksDisp4(N/4, M/4); 
	dim3 threadsPerBlockDisp8(64, 8);
	dim3 numbBlocksDisp8(N/8, M/8); 
	dim3 threadsPerBlockDisp12(64, 12);
	dim3 numbBlocksDisp12(N/12, M/12); 
	dim3 threadsPerBlockDisp16(64, 16);
	dim3 numbBlocksDisp16(N/16, M/16); 
	dim3 threadsPerBlockDisp32(64, 16);
	dim3 numbBlocksDisp32(N/32, M/32); 	
	
	edgeMatch2<<<numbBlocksDisp2, threadsPerBlockDisp2>>>( rows, cols, g_srcL, g_srcR, g_match_2, 0, 0);
	edgeMatch16<<<numbBlocksDisp16, threadsPerBlockDisp16, 64*16*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_16, 0, 0);
	edgeMatch32<<<numbBlocksDisp32, threadsPerBlockDisp32, 64*32*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_32, 0, 0);

	edgeMatch2<<<numbBlocksDisp2, threadsPerBlockDisp2>>>( rows, cols, g_srcL, g_srcR, g_match_2x, 1, 0);
	edgeMatch16<<<numbBlocksDisp16, threadsPerBlockDisp16, 64*16*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_16x, 8, 0);
	edgeMatch32<<<numbBlocksDisp32, threadsPerBlockDisp32, 64*32*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_32x, 16, 0);
	
	edgeMatch2<<<numbBlocksDisp2, threadsPerBlockDisp2>>>( rows, cols, g_srcL, g_srcR, g_match_2y, 0, 1);
	edgeMatch16<<<numbBlocksDisp16, threadsPerBlockDisp16, 64*16*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_16y, 0, 8);
	edgeMatch32<<<numbBlocksDisp32, threadsPerBlockDisp32, 64*32*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_32y, 0, 16);
	
	edgeMatch2<<<numbBlocksDisp2, threadsPerBlockDisp2>>>( rows, cols, g_srcL, g_srcR, g_match_2xy, 1, 1);
	edgeMatch16<<<numbBlocksDisp16, threadsPerBlockDisp16, 64*16*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_16xy, 8, 8);
	edgeMatch32<<<numbBlocksDisp32, threadsPerBlockDisp32, 64*32*sizeof(unsigned int)>>>( rows, cols, g_srcL, g_srcR, g_match_32xy, 16, 16);
 	hipDeviceSynchronize();
	
	dim3 threadsPerBlockDisp64(1, 1, 64);
	dim3 numbBlocksDisp64(N/1, M/1, 1); 
	brain1<<<numbBlocksDisp64, threadsPerBlockDisp64>>>( rows, cols, g_R_edge, g_match_2, g_match_4, g_match_8, g_match_16, g_match_32,  g_match_2x, g_match_4x, g_match_8x, g_match_16x, g_match_32x, g_match_2y, g_match_4y, g_match_8y, g_match_16y, g_match_32y, g_match_2xy, g_match_4xy, g_match_8xy, g_match_16xy, g_match_32xy, g_params, g_edgeMached);
	

	dim3 threadsPerBlock4(4, 4);
	dim3 numbBlocks4(N/ threadsPerBlock4.x,M/ threadsPerBlock4.y); 
	//median<<<numbBlocks4, threadsPerBlock4>>>(rows, cols, g_R_edge, g_L_edge, g_disp);
	
	hipDeviceSynchronize();
	median5x5Edge<<<numbBlocks4, threadsPerBlock4>>>(rows, cols, g_L_edge, g_edgeMached,  g_disp);
	hipDeviceSynchronize();
	
}*/

unsigned char** cv::gpu::mj::initDisp3C(const int size){
	
	unsigned char** temps = (unsigned char**)malloc(3*sizeof(unsigned char*));

	for(int i = 0; i < 3; i++){
		hipMalloc((void **)&(temps[i]), size);
	} 
	return temps;
}
__global__ void fillInitParamsDisp3(unsigned short * t){
	if( threadIdx.x == 0 &&  threadIdx.y == 0){ 
	int multi = 3;
		t[0 ] = multi*45; 
		t[1 ] = multi*40; 
		t[2 ] = multi*35; 
		t[3 ] = multi*30;
		t[4 ] = multi*29; 
		t[5 ] = multi*28; 
		t[6 ] = multi*25; 
		t[7 ] = multi*20;
		t[8 ] = multi*18; 
		t[9 ] = multi*16; 
		t[10] = multi*14; 
		t[11] = multi*12;
		t[12] = multi*11; 
		t[13] = multi*10; 
		t[14] = multi*9; 
		t[15] = multi*8;
		t[16] = multi*7; 
		t[17] = multi*6; 
		t[18] = multi*5; 
		t[19] = multi*4;
		t[20] = multi*3; 
		t[21] = multi*2; 
		t[22] = multi*1; 
		t[23] = multi*1; 
		t[24] = multi*1; 
	}

}
unsigned short** cv::gpu::mj::initDisp3US(const int size){
	
	unsigned short** temps = (unsigned short**)malloc(2*sizeof(unsigned short*));
	
	hipMalloc((void **)&(temps[0]), size/(8*8)*64);
	hipMalloc((void **)&(temps[1]), 32*sizeof(unsigned short));
	hipDeviceSynchronize();
	unsigned short * t = temps[1];
	
	fillInitParamsDisp3<<<1, 1>>>(t);
	
	return temps;
}


void cv::gpu::mj::disp3(const int rows,const int cols, unsigned char *g_srcL, unsigned char *g_srcR, unsigned char* g_disp, unsigned char** tempsC, unsigned short** tempsUS ){
	
	int w = WIDTH;
	int h = HEIGHT;
	
	size_t offset = 0;
	
	unsigned char* g_L_edge = tempsC[0];
	unsigned char* g_R_edge = tempsC[1];
	//unsigned char* g_udisp = tempsC[2];
	
	unsigned short* g_match_8 = tempsUS[0];
	unsigned short* g_w = tempsUS[1];
	
	const int size = sizeof(unsigned char)*rows*cols;
	
	hipMemset(g_disp, 0, size);
	hipMemset(g_L_edge, 0, size);
	hipMemset(g_R_edge, 0, size);
    hipBindTexture2D(&offset, tex2Dleft,  g_srcL, ca_desc0, w, h, w*4);
    hipBindTexture2D(&offset, tex2Dright, g_srcR, ca_desc1, w, h, w*4);
    
	hipDeviceSynchronize();
	dim3 threadsPerBlock(16, 16);
	dim3 numbBlocks(w/ threadsPerBlock.x,h/ threadsPerBlock.y); 
	edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcL, g_L_edge, 50);
	hipDeviceSynchronize();
	dim3 threadsPerBlock24(24, 24);
	dim3 numbBlocks24(w/ threadsPerBlock24.x,h/ threadsPerBlock24.y); 
	findDistanceFast<<<numbBlocks24, threadsPerBlock24,64*64*sizeof(unsigned char) >>>(rows, cols, g_L_edge, g_R_edge);
	hipDeviceSynchronize();
	dim3 threadsPerBlockDisp16(64, 16);
	dim3 numbBlocksDisp16(w/16, h/16); 
	edgeMatch8w16<<<numbBlocksDisp16,threadsPerBlockDisp16,16*64*2*sizeof(unsigned short)>>>(rows, cols, g_srcL, g_srcR, g_match_8);	
 	dim3 threadsPerBlockDisp64(16, 16, 2);
	dim3 numbBlocksDisp64(w/16, h/16, 1); 
 	int extSize = 3*3*64*sizeof(unsigned int)+6*6*64*sizeof(unsigned short)+16*16*64*sizeof(unsigned short)+4*4*64*sizeof(unsigned int)+4*4*32*sizeof(unsigned char)+4*4*32*sizeof(unsigned int);
 	brain3<<<numbBlocksDisp64, threadsPerBlockDisp64, extSize>>>( rows, cols, g_srcL, g_srcR, g_R_edge, g_match_8, g_w, g_disp, 1, 1000*256);

 	hipUnbindTexture(tex2Dleft);
 	hipUnbindTexture(tex2Dright);
 	//perform Extended 2x2 maching 
 	//
	//	1|2|1
	//	2|4|2
	//	1|2|1
	//
	// and mix it with proportional 16 and 32 block 
	
}
void cv::gpu::mj::dispToUdepth(const int rows,const int cols, const int uRows,const int uCols, unsigned char *g_disp, unsigned char *g_udepth, unsigned char** tempsC){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* g_udisp = tempsC[0];
	const int size = sizeof(unsigned char)*rows*cols;
	hipMemset(g_udisp, 0, size);
	 	
	dim3 threadsPerBlockUDisp(1, 480);
	dim3 numbBlocksUDisp(N/1, M/480); 
	udisp<<<threadsPerBlockUDisp, numbBlocksUDisp>>>(rows, cols, g_disp, g_udisp);
	hipDeviceSynchronize();
	hipMemset(g_udepth, 0, size);
	dim3 threadsPerBlockUDepth(1, 256);
	dim3 numbBlocksUDepth(N/1, M/256);
	udispToUdepth<<<threadsPerBlockUDepth,numbBlocksUDepth>>>(uRows, uCols, g_udisp, g_udepth);
}




void cv::gpu::mj::rectif(const int rows,const int cols, unsigned char *srcL, unsigned char *srcR, unsigned char *dstL, unsigned char *dstR, unsigned char * out){
	
	
}

void cv::gpu::mj::realocHostMem(int sizec, unsigned char *img){
	unsigned char* cpudataSrc;
	hipHostMalloc 	((void **)&cpudataSrc,sizeof(unsigned char));
	//hipMemcpy(cpudataSrc, img, sizec, hipMemcpyHostToHost);
	//img = cpudataSrc;		
}

void cv::gpu::mj::cudaMemAlocImagePtr(unsigned char *dest, int size){
	//hipSetDeviceFlags(hipDeviceMapHost);
	hipHostMalloc 	((void **)&dest, size);
	//hipHostAlloc((void **)&dest,  size,  hipHostMallocMapped);
}

void cv::gpu::mj::cudaMemcpyHtoH(unsigned char *src, unsigned char *dest, int size){
	//hipHostMalloc 	((void **)&dest,size);
	hipMemcpy(dest, src, size, hipMemcpyHostToHost);	
}

void cv::gpu::mj::cudaMemcpyHtoD(unsigned char *src, unsigned char* dest, int size){

	hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

void cv::gpu::mj::cudaMemcpyDtoH(unsigned char *src, unsigned char* dest, int size){

	hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
}

void cv::gpu::mj::cudaInit(unsigned char** g_src1, unsigned char** g_src2, unsigned char** g_disp, const int rows, const int cols){
	
	size_t offset = 0;
    
	const int size = sizeof(unsigned char)*rows*cols;

	hipMalloc((void **)g_src1, size);
	hipMalloc((void **)g_src2, size);
	hipMalloc((void **)g_disp, size);
	
	tex2Dleft.addressMode[0] = hipAddressModeClamp;
    tex2Dleft.addressMode[1] = hipAddressModeClamp;
    tex2Dleft.filterMode     = hipFilterModePoint;
    tex2Dleft.normalized     = false;
    tex2Dright.addressMode[0] = hipAddressModeClamp;
    tex2Dright.addressMode[1] = hipAddressModeClamp;
    tex2Dright.filterMode     = hipFilterModePoint;
    tex2Dright.normalized     = false;

	
}

void cv::gpu::mj::cudaDestroy(unsigned char* g_src1, unsigned char* g_src2, unsigned char* g_disp, unsigned char** g_temps){
	
	hipFree(g_src1);
	hipFree(g_src2);
	hipFree(g_disp);
	int i;
	for(i = 0; i < 8; i++){
		hipFree(g_temps[i]);
	} 
}

void cv::gpu::mj::cudaDestroyDisp2(unsigned char* g_src1, unsigned char* g_src2, unsigned char* g_disp, unsigned char** g_tempsC, unsigned int** g_tempsI){
	
	hipFree(g_src1);
	hipFree(g_src2);
	hipFree(g_disp);
	int i;
	for(i = 0; i < 3; i++){
		hipFree(g_tempsC[i]);
	} 
	for(i = 0; i < 20; i++){
		hipFree(g_tempsI[i]);
	} 
}

void cv::gpu::mj::cudaDestroyDisp3(unsigned char* g_src1, unsigned char* g_src2, unsigned char* g_disp, unsigned char** g_tempsC, unsigned short** g_tempsUC){
	
	hipFree(g_src1);
	hipFree(g_src2);
	hipFree(g_disp);
	int i;
	for(i = 0; i < 3; i++){
		hipFree(g_tempsC[i]);
	} 
	for(i = 0; i < 2; i++){
		hipFree(g_tempsUC[i]);
	} 
}




























