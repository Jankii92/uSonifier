#include "hip/hip_runtime.h"
#include "improc.h"
#include <stdio.h>
#include <chrono>

#include <iostream>

#define PI 3.14159265
#define NUM_STREAMS 2
#define WIDTH	640
#define HEIGHT	480


using namespace std;



void cv::gpu::mj::blur(const int rows,const int cols, const int k, unsigned char *src, unsigned char* dst){
	
	auto start1 = std::chrono::system_clock::now();
	int N = WIDTH;
	int M = HEIGHT;
	//hipSetDeviceFlags(hipDeviceMapHost);
	
	unsigned char* gpudataSrc;
	unsigned char* gpudataOut;
	unsigned char* cpudataOut;
	
	int size = sizeof(unsigned char)*rows*cols;
	
	hipHostMalloc 	((void **)&cpudataOut,size);	
	
	auto stop1 = std::chrono::system_clock::now();
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	//hipHostGetDevicePointer((void **)&gpudataOut,  (void *) dst , 0);
	hipMemcpy(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	auto stop2 = std::chrono::system_clock::now();
	dim3 threadsPerBlock(16,16);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	//auto start1 = std::chrono::system_clock::now();
	//hipDeviceSynchronize();
	auto stop3 = std::chrono::system_clock::now();
	//blur_noShare_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut );
	//hipDeviceSynchronize();
	auto stop4 = std::chrono::system_clock::now();
	//hipDeviceSynchronize();
	cout<<"1: Done!!!"<<endl;
	//blur_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut);
	cout<<"3: Done!!!"<<endl;
	//hipDeviceSynchronize();
	auto stop5 = std::chrono::system_clock::now();
	hipMemcpy(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
	auto duration1 = (std::chrono::duration_cast<std::chrono::microseconds>(stop1 - start1).count());
	auto duration2 = (std::chrono::duration_cast<std::chrono::microseconds>(stop2 - stop1).count());
	auto duration3 = (std::chrono::duration_cast<std::chrono::microseconds>(stop3 - stop2).count());
	auto duration4 = (std::chrono::duration_cast<std::chrono::microseconds>(stop4 - stop3).count());
	auto duration5 = (std::chrono::duration_cast<std::chrono::microseconds>(stop5 - stop4).count());
	auto duration =(std::chrono::duration_cast<std::chrono::microseconds>(stop5 - start1).count());
	
    //cout<<(int)duration1<<" "<<(int)duration2<<" " <<(int)duration3<<" " <<(int)duration4<<" " <<(int)duration5<<(int)duration6<< endl;
}
void cv::gpu::mj::sobel(const int rows,const int cols, unsigned char *src, unsigned char* dst, int mode){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* gpudataSrc;
	
	unsigned char* gpudataOut;
	
	const int size = sizeof(unsigned char)*rows*cols;
		
	
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	
	hipMemcpyAsync(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(32,32);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	

	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataSrc, gpudataOut, mode);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
}


void cv::gpu::mj::disp(const int rows,const int cols, unsigned char *g_srcL, unsigned char *g_srcR, unsigned char* g_disp, int shift){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	//int* g_tmpL;
	//unsigned char* g_tmpOut;
	unsigned char* g_tmpL;
	unsigned char* g_tmpR;
	unsigned char* g_tmpL2;
	unsigned char* g_tmpR2;
	
	const int size = sizeof(unsigned char)*rows*cols;
	
	hipMalloc((void **)&g_tmpL, size);
	hipMalloc((void **)&g_tmpR, size);
	hipMalloc((void **)&g_tmpL2, size);
	hipMalloc((void **)&g_tmpR2, size);
	//hipMalloc((void **)&g_tmpOut, size);
	//hipMalloc((void **)&g_tmpOut, size);
	//hipMalloc((void **)&g_tmp3L, size);
	hipMemset(g_disp, 0, size);
	//hipMemcpyAsync(g_srcR, srcR, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(16, 16);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	//hipDeviceSynchronize();
	//cout<<"1: Done!!!"<<endl;
	//ctoi_GPU<<<numbBlocks, threadsPerBlock>>>(rows , cols, g_srcL, g_tmpL);
  	//hipDeviceSynchronize();
  	
	//prewittFS_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpLX, g_tmpLY, g_tmp2LX, g_tmp2LY);
	
	edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcL, g_tmpL, 50);
	edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcR, g_tmpR, 50);
	//hipDeviceSynchronize();
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpL2);
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpR, g_tmpR2);
	hipDeviceSynchronize();
	reduce<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL2, g_tmpL);
	reduce<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpR2, g_tmpR);
	hipDeviceSynchronize();
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpL2);
	edgeTypeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpR, g_tmpR2);
	hipDeviceSynchronize();
	
	
	dim3 threadsPerBlockDisp(8, 8);
	dim3 numbBlocksDisp(N/ threadsPerBlockDisp.x,M/ threadsPerBlockDisp.y); 
	compare<<<numbBlocksDisp, threadsPerBlockDisp>>>(rows, cols, g_tmpL2, g_tmpR2, g_disp, shift);
	//blend_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpR, g_disp, 0.5, 1);
	//findNode<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_disp, g_disp);
	
	//dim1 threadsPerBlock1(1);
	//dim1 numbBlocks1(1024/ threadsPerBlock.x); 
	
	//edgeDraw<<<200, 1>>>(rows, cols, g_tmpOut, g_disp);
	
  	//hipDeviceSynchronize();
	//edgeDetect_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_srcR, g_srcR, 30);
  	/*prewittX_GPU <<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpLX, 0);
  	prewittY_GPU <<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmpLY, 0);
  	prewittXsec_GPU <<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmp2LX, 0);
  	prewittYsec_GPU <<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpL, g_tmp2LY, 0);
  	hipDeviceSynchronize();
  	*/
  	
	///edgeDetect<<<numbBlocks, threadsPerBlock>>>(rows, cols, g_tmpLX, g_tmpLY, g_tmp2LX, g_tmp2LY, g_tmpOut);
	
	//itoc_GPU<<<numbBlocks, threadsPerBlock>>>(rows , cols, g_tmpOut, g_disp);
  	//hipDeviceSynchronize();
	//hipFree(g_tmpOut);
	hipFree(g_tmpL);
	hipFree(g_tmpR);
	hipFree(g_tmpL2);
	hipFree(g_tmpL2);
}


void cv::gpu::mj::rectif(const int rows,const int cols, unsigned char *srcL, unsigned char *srcR, unsigned char *dstL, unsigned char *dstR, unsigned char * out){
	
	
}

void cv::gpu::mj::realocHostMem(int sizec, unsigned char *img){
	unsigned char* cpudataSrc;
	hipHostMalloc 	((void **)&cpudataSrc,sizeof(unsigned char));
	//hipMemcpy(cpudataSrc, img, sizec, hipMemcpyHostToHost);
	//img = cpudataSrc;		
}

void cv::gpu::mj::cudaMemAlocImagePtr(unsigned char *dest, int size){
	//hipSetDeviceFlags(hipDeviceMapHost);
	hipHostMalloc 	((void **)&dest, size);
	//hipHostAlloc((void **)&dest,  size,  hipHostMallocMapped);
}

void cv::gpu::mj::cudaMemcpyHtoH(unsigned char *src, unsigned char *dest, int size){
	//hipHostMalloc 	((void **)&dest,size);
	hipMemcpy(dest, src, size, hipMemcpyHostToHost);	
}

void cv::gpu::mj::cudaMemcpyHtoD(unsigned char *src, unsigned char* dest, int size){

	hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

void cv::gpu::mj::cudaMemcpyDtoH(unsigned char *src, unsigned char* dest, int size){

	hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
}

void cv::gpu::mj::cudaInit(unsigned char** g_src1, unsigned char** g_src2, unsigned char** g_disp, const int rows, const int cols){
	
	const int size = sizeof(unsigned char)*rows*cols;

	hipMalloc((void **)g_src1, size);
	hipMalloc((void **)g_src2, size);
	hipMalloc((void **)g_disp, size);
	
}

void cv::gpu::mj::cudaDestroy(unsigned char* g_src1, unsigned char* g_src2, unsigned char* g_disp){
	
	hipFree(g_src1);
	hipFree(g_src2);
	hipFree(g_disp);
}




























