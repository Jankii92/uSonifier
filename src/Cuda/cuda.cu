#include "hip/hip_runtime.h"
#include "improc.h"
#include <stdio.h>
#include <chrono>

#include <iostream>

#define PI 3.14159265
#define WIDTH	640
#define HEIGHT	480


using namespace std;


int calcSum(unsigned char *src){
	
	int xh = 480/2;
	int yh = 640/2;
	int sum = 0;
	for( int x = xh-50; x < xh+50 ; x++){
		for( int y = yh-50; y < yh+50 ; y++){
			sum+=src[y*640+x];
		}
	}
	return sum;


}


void cv::gpu::mj::blur(const int rows,const int cols, const int k, const unsigned char *src, unsigned char* dst){
	
	auto start1 = std::chrono::system_clock::now();
	int N = WIDTH;
	int M = HEIGHT;
	//hipSetDeviceFlags(hipDeviceMapHost);
	
	unsigned char* gpudataSrc;
	unsigned char* gpudataOut;
	unsigned char* cpudataOut;
	
	int size = sizeof(unsigned char)*rows*cols;
	
	hipHostMalloc 	((void **)&cpudataOut,size);	
	
	auto stop1 = std::chrono::system_clock::now();
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	//hipHostGetDevicePointer((void **)&gpudataOut,  (void *) dst , 0);
	hipMemcpyAsync(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	auto stop2 = std::chrono::system_clock::now();
	dim3 threadsPerBlock(32,32);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	//auto start1 = std::chrono::system_clock::now();
	auto stop3 = std::chrono::system_clock::now();
	blur_noShare_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut );
	//blur_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, k, gpudataSrc, gpudataOut);
	auto stop4 = std::chrono::system_clock::now();
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	auto stop5 = std::chrono::system_clock::now();
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
	auto duration1 = std::chrono::duration_cast<std::chrono::microseconds>(stop1 - start1);
	auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(stop2 - stop1);
	auto duration3 = std::chrono::duration_cast<std::chrono::microseconds>(stop3 - stop2);
	auto duration4 = std::chrono::duration_cast<std::chrono::microseconds>(stop4 - stop3);
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop5 - start1);
	
    //cout<<(int)duration1.count()<<" "<<(int)duration2.count()<<" " <<(int)duration3.count()<<" " <<(int)duration4.count()<<" " <<(int)duration.count()<< endl;
}
void cv::gpu::mj::sobel(const int rows,const int cols, const unsigned char *src, unsigned char* dst, int mode){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* gpudataSrc;
	
	unsigned char* gpudataMid;
	unsigned char* gpudataOut;
	
	const int size = sizeof(unsigned char)*rows*cols;
		
	
	hipMalloc((void **)&gpudataSrc, size);
	hipMalloc((void **)&gpudataOut, size);
	hipMalloc((void **)&gpudataMid, size);
	
	hipMemcpyAsync(gpudataSrc, src, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(32,32);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	
	cout<<"MAT!!!!!!!"<<endl;

	float angle = -2.0f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = -1.5f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = 1.0f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = 0.5f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = 0.0f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = 0.5f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	angle = 0.0f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrc, gpudataMid, angle*PI/180);
	sobel_abs_GPU<<<numbBlocks, threadsPerBlock>>>(rows,cols, gpudataMid, gpudataOut, 1);
	hipMemcpyAsync(dst, gpudataOut, size, hipMemcpyDeviceToHost);
	cout<<"Angle:"<<angle<<" Sum:"<<calcSum(dst)<<endl;
	
	hipFree(gpudataMid);
	hipFree(gpudataSrc);
	hipFree(gpudataOut);
}

void cv::gpu::mj::rectif(const int rows,const int cols, const unsigned char *srcL, const unsigned char *srcR, unsigned char *dstL, unsigned char *dstR, unsigned char * out){
	
	int N = WIDTH;
	int M = HEIGHT;
	
	unsigned char* gpudataSrcL;
	unsigned char* gpudataSrcR;
	unsigned char* gpudataOutL;
	unsigned char* gpudataOutR;
	unsigned char* gpudataOut;
		
	const int size = sizeof(unsigned char)*rows*cols;
		
	
	hipMalloc((void **)&gpudataSrcL, size);
	hipMalloc((void **)&gpudataOutL, size);
	hipMalloc((void **)&gpudataSrcR, size);
	hipMalloc((void **)&gpudataOutR, size);
	hipMalloc((void **)&gpudataOut, size);
	
	hipMemcpy(gpudataSrcL, srcL, size, hipMemcpyHostToDevice);
	hipMemcpy(gpudataSrcR, srcR, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(32,32);
	dim3 numbBlocks(N/ threadsPerBlock.x,M/ threadsPerBlock.y); 
	
	//cout<<"MAT!!!!!!!"<<endl;

	float angle = 0.0f;
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrcR, gpudataOutR, 0);
	rotate_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols, gpudataSrcL, gpudataOutL, angle*PI/180);
	hipMemcpyAsync(dstL, gpudataOutL, size, hipMemcpyDeviceToHost);
	hipMemcpyAsync(dstR, gpudataOutR, size, hipMemcpyDeviceToHost);

	blend_GPU<<<numbBlocks, threadsPerBlock>>>(rows, cols,  gpudataOutL,  gpudataOutR, gpudataOut, 0.5f);
	
	hipMemcpy(out, gpudataOut, size, hipMemcpyDeviceToHost);
	hipFree(gpudataSrcL);
	hipFree(gpudataOutL);
	hipFree(gpudataSrcR);
	hipFree(gpudataOutR);
	hipFree(gpudataOut);
}



void cv::gpu::mj::realocHostMem(int sizec, unsigned char *img){
	unsigned char* cpudataSrc;
	hipHostMalloc 	((void **)&cpudataSrc,sizeof(unsigned char));
	//hipMemcpy(cpudataSrc, img, sizec, hipMemcpyHostToHost);
	//img = cpudataSrc;		
}

void cv::gpu::mj::cudaMemAlocImagePtr(unsigned char *dest, int size){
	//hipSetDeviceFlags(hipDeviceMapHost);
	hipHostMalloc 	((void **)&dest, size);
	//hipHostAlloc((void **)&dest,  size,  hipHostMallocMapped);
}

void cv::gpu::mj::cudaMemcpyHtoH(unsigned char *src, unsigned char *dest, int size){
	//hipHostMalloc 	((void **)&dest,size);
	hipMemcpy(dest, src, size, hipMemcpyHostToHost);	
}
























